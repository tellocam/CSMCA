#include "hip/hip_runtime.h"
#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>


// result = (x, y)
__global__ void cuda_1c(int N, double *x, double *sum, double *abssum, double *squares, double *zeros)
{

  double sum_thr = 0;
  double abssum_thr = 0;
  double squares_thr = 0;
  double zeros_thr = 0;

  //calculation
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    sum_thr  += x[i];
    abssum_thr += abs(x[i]);
    squares_thr += pow(x[i],2);
    zeros_thr += (double)x[i]==0;
  }
  //reduction
  for (int i=warpSize/2; i>0; i=i/2){
     sum_thr += __shfl_down_sync(-1, sum_thr, i);
     abssum_thr += __shfl_down_sync(-1, abssum_thr, i);
     squares_thr += __shfl_down_sync(-1, squares_thr, i);
     zeros_thr += __shfl_down_sync(-1, zeros_thr, i);
  }

  //atomicAdd

  if ((threadIdx.x &(warpSize-1))== 0) {
    atomicAdd(sum, sum_thr);
    atomicAdd(abssum, abssum_thr);
    atomicAdd(squares, squares_thr);
    atomicAdd(zeros, zeros_thr);
    
  }


}



int main() {

  int N = 1000000;

  // Allocate arrays and doubles on CPU
  double *x = (double *)malloc(sizeof(double) * N);
  double *x_sum = (double *)malloc(sizeof(double));
  double *x_abssum = (double *)malloc(sizeof(double));
  double *x_squares = (double *)malloc(sizeof(double));
  double *x_zeros = (double *)malloc(sizeof(double));
  // Initialize arrays and doubles on CPU
  std::fill(x, x + N, 1);
  *x_sum = 0;
  *x_abssum = 0;
  *x_squares = 0;
  *x_zeros = 0;

  // Allocate arrays and doubles on GPU
  double *cuda_x, *cuda_sum, *cuda_abssum, *cuda_squares, *cuda_zeros;
  
  CUDA_ERRCHK(hipMalloc(&cuda_x, sizeof(double) * N));
  CUDA_ERRCHK(hipMalloc(&cuda_sum, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_abssum, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_squares, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_zeros, sizeof(double)));

  // Initialize arrays and doubles on GPU
  CUDA_ERRCHK(hipMemcpy(cuda_x, x, sizeof(double) * N, hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_sum, x_sum, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_abssum, x_abssum, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_squares, x_squares, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_zeros, x_zeros, sizeof(double), hipMemcpyHostToDevice));

  cuda_1c<<<((N+255)/256), 256>>>(N, cuda_x, cuda_sum, cuda_abssum, cuda_squares, cuda_zeros);
  
  CUDA_ERRCHK(hipMemcpy(x_sum, cuda_sum, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(x_abssum, cuda_abssum, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(x_squares, cuda_squares, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(x_zeros, cuda_zeros, sizeof(double), hipMemcpyDeviceToHost));

  std::cout << "Vector X 1-Norm: " << *x_sum << std::endl;
  std::cout << "Vector X Absolute Sum: " << *x_abssum << std::endl;
  std::cout << "Vector X 2-Norm: " << *x_squares << std::endl;
  std::cout << "Vector X Zeros: " << *x_zeros << std::endl;

  //
  // Clean up
  //
  CUDA_ERRCHK(hipFree(cuda_x));
  CUDA_ERRCHK(hipFree(cuda_sum));
  CUDA_ERRCHK(hipFree(cuda_abssum));
  CUDA_ERRCHK(hipFree(cuda_squares));
  CUDA_ERRCHK(hipFree(cuda_zeros));
  free(x);
  free(x_sum);
  free(x_abssum);
  free(x_squares);
  free(x_zeros);

  return EXIT_SUCCESS;
}