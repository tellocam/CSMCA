#include "hip/hip_runtime.h"
#include "timer.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <vector>

__global__ void cuda_5_1c(double *x, double *y, double *z, int N)
{
  unsigned int total_threads = blockDim.x * gridDim.x;
	unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned int i = global_tid; i<N; i += total_threads) {
		z[i] = x[i] + y[i];
	}

}

double findMedian(std::vector<double> a,
                  int n)
{
  if (n % 2 == 0) {
      std::nth_element(a.begin(),
                  a.begin() + n / 2,
                  a.end());
      std::nth_element(a.begin(),
                  a.begin() + (n - 1) / 2,
                  a.end());
      return (double)(a[(n - 1) / 2]
                      + a[n / 2])
             / 2.0;
  }
  else {
      std::nth_element(a.begin(),
                  a.begin() + n / 2,
                  a.end());
      return (double)a[n / 2];
    }
}

void fillVectorWithIndices(double *V, int N){
  for (size_t i=0; i<N; ++i){
    V[i] = i;
  }
}

int main() {

int median_int = 20; // iterations to build median for timings
int N_max = 28; // 2 to the N_max'th power is the largest vector for the bandwidth measurement

Timer timer;
std::vector<double> timings, peak_bw, N_vec;
for (size_t i=0; i<=N_max; ++i){
  N_vec.push_back(pow(2,i));
}

double *x, *y, *z, *gpu_x, *gpu_y, *gpu_z;

std::cout << "Effective Bandwidth for varying Vector size addition: " << std::endl;
std::cout << "N, " << "GB/s" << std::endl;


//First For Loop to obtain Timings for varying N
for (size_t i=0; i<N_max; ++i){

  //prepare data on GPU and CPU for Kernel Submission
  x = (double*)malloc(N_vec[i]*sizeof(double));
  y = (double*)malloc(N_vec[i]*sizeof(double));
	z = (double*)malloc(N_vec[i]*sizeof(double));
  fillVectorWithIndices(x,N_vec[i]);
  fillVectorWithIndices(y,N_vec[i]);
  // std::fill(x, x + (int)N_vec[i], 1);
  // std::fill(y, y + (int)N_vec[i], 1);
  std::fill(z, z + (int)N_vec[i], 0);
  hipMalloc(&gpu_x, N_vec[i]*sizeof(double));
  hipMalloc(&gpu_y, N_vec[i]*sizeof(double));
  hipMalloc(&gpu_z, N_vec[i]*sizeof(double));
  hipMemcpy(gpu_x, x, N_vec[i]*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_y, x, N_vec[i]*sizeof(double), hipMemcpyHostToDevice);
  // hipMemcpy(gpu_z, z, N_vec[i]*sizeof(double), hipMemcpyHostToDevice);


  //First nested for Loop to obtain median of N_max[i]
  for(int j=0; j < median_int; j++){
    hipDeviceSynchronize();
    timer.reset();
    cuda_5_1c<<<((N_vec[i]+255)/256), 256>>>(gpu_x, gpu_y, gpu_z, N_vec[i]);
    hipDeviceSynchronize();
    timings.push_back(timer.get());
  }

  hipMemcpy(z, gpu_z, N_vec[i]*sizeof(double), hipMemcpyDeviceToHost);

  // obtain median timing for N_vec[i] from all timings_int iterations and clear timings vector for N_vec[i+1]
  // 3 * floor((N - k_values[i])) * sizeof(double) * pow(10, -9) / findMedian(exec_timings, 10);
  peak_bw.push_back((3*N_vec[i]*sizeof(double)*pow(10,-9))/findMedian(timings, median_int));
  timings.clear();

  // print N[i] and peak_bw[i] for copying it into csv later on :-)

  std::cout << N_vec[i] << ", " << peak_bw[i] << std::endl;

  hipFree(gpu_x);
  hipFree(gpu_y);
  hipFree(gpu_z);
  free(x);
  free(y);
  free(z);

}
return EXIT_SUCCESS;
}

