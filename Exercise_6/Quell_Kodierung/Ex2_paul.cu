#include "hip/hip_runtime.h"
#include "poisson2d.hpp"
#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <fstream>
 
// y = A * x
__global__ void cuda_csr_matvec_product(int N, int *csr_rowoffsets,
                                        int *csr_colindices, double *csr_values,
                                        double *x, double *y)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    double sum = 0;
    for (int k = csr_rowoffsets[i]; k < csr_rowoffsets[i + 1]; k++) {
      sum += csr_values[k] * x[csr_colindices[k]];
    }
    y[i] = sum;
  }
}
 
// x <- x + alpha * y
__global__ void cuda_vecadd(int N, double *x, double *y, double alpha)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    x[i] += alpha * y[i];
}
 
// x <- y + alpha * x
__global__ void cuda_vecadd2(int N, double *x, double *y, double alpha)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    x[i] = y[i] + alpha * x[i];
}
 
// result = (x, y)
__global__ void cuda_dot_product(int N, double *x, double *y, double *result)
{
  __shared__ double shared_mem[512];
 
  double dot = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    dot += x[i] * y[i];
  }
 
  shared_mem[threadIdx.x] = dot;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[threadIdx.x] += shared_mem[threadIdx.x + k];
    }
  }
 
  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}
 
 
__global__ void count_nnz(int *row_offsets, int N, int M) {
  for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < N*M; row += gridDim.x * blockDim.x) {
    int nnz_for_this_node = 1;
    int i = row / N;
    int j = row % N;
    if (i > 0) nnz_for_this_node += 1;
    if (j > 0) nnz_for_this_node += 1;
    if (i < N-1) nnz_for_this_node += 1;
    if (j < M-1) nnz_for_this_node += 1;
    row_offsets[row] = nnz_for_this_node;
  }
}
 
__global__ void scan_kernel_1(int const *X,
                              int *Y,
                              int N,
                              int *carries)
{
  __shared__ int shared_buffer[256];
  int my_value;
 
  unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
  unsigned int block_start = work_per_thread * blockDim.x *  blockIdx.x;
  unsigned int block_stop  = work_per_thread * blockDim.x * (blockIdx.x + 1);
  unsigned int block_offset = 0;
 
  // run scan on each section
  for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
  {
    // load data:
    my_value = (i < N) ? X[i] : 0;
 
    // inclusive scan in shared buffer:
    for(unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
      __syncthreads();
      shared_buffer[threadIdx.x] = my_value;
      __syncthreads();
      if (threadIdx.x >= stride)
        my_value += shared_buffer[threadIdx.x - stride];
    }
    __syncthreads();
    shared_buffer[threadIdx.x] = my_value;
    __syncthreads();
 
    // exclusive scan requires us to write a zero value at the beginning of each block
    my_value = (threadIdx.x > 0) ? shared_buffer[threadIdx.x - 1] : 0;
 
    // write to output array
    if (i < N)
      Y[i] = block_offset + my_value;
 
    block_offset += shared_buffer[blockDim.x-1];
  }
 
  // write carry:
  if (threadIdx.x == 0)
    carries[blockIdx.x] = block_offset;
 
}
 
// exclusive-scan of carries
__global__ void scan_kernel_2(int *carries)
{
  __shared__ int shared_buffer[256];
 
  // load data:
  int my_carry = carries[threadIdx.x];
 
  // exclusive scan in shared buffer:
 
  for(unsigned int stride = 1; stride < blockDim.x; stride *= 2)
  {
    __syncthreads();
    shared_buffer[threadIdx.x] = my_carry;
    __syncthreads();
    if (threadIdx.x >= stride)
      my_carry += shared_buffer[threadIdx.x - stride];
  }
  __syncthreads();
  shared_buffer[threadIdx.x] = my_carry;
  __syncthreads();
 
  // write to output array
  carries[threadIdx.x] = (threadIdx.x > 0) ? shared_buffer[threadIdx.x - 1] : 0;
}
 
__global__ void scan_kernel_3(int *Y, int N,
                              int const *carries)
{
  unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
  unsigned int block_start = work_per_thread * blockDim.x *  blockIdx.x;
  unsigned int block_stop  = work_per_thread * blockDim.x * (blockIdx.x + 1);
 
  __shared__ int shared_offset;
 
  if (threadIdx.x == 0)
    shared_offset = carries[blockIdx.x];
 
  __syncthreads();
 
  // add offset to each element in the block:
  for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
    if (i < N)
      Y[i] += shared_offset;
}
 
void exclusive_scan(int const * input,
                    int       * output, int N)
{
  int num_blocks = 256;
  int threads_per_block = 256;
 
  int *carries;
  hipMalloc(&carries, sizeof(int) * num_blocks);
 
  // First step: Scan within each thread group and write carries
  scan_kernel_1<<<num_blocks, threads_per_block>>>(input, output, N, carries);
 
  // Second step: Compute offset for each thread group (exclusive scan for each thread group)
  scan_kernel_2<<<1, num_blocks>>>(carries);
 
  // Third step: Offset each thread group accordingly
  scan_kernel_3<<<num_blocks, threads_per_block>>>(output, N, carries);
 
  hipFree(carries);
}
 
__global__ void assembleA(int *row_offsets, int N, int M, int *col_indices, double *values) {
  for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < N*M; row += gridDim.x * blockDim.x) {
    int i = row / N; // row of node
    int j = row % N; // col of node
    int this_row_offset = row_offsets[row];
    // diagonal entry
    col_indices[this_row_offset] = i * N + j;
    values[this_row_offset] = 4;
    this_row_offset += 1;
    if (i > 0) { // bottom neighbor
      col_indices[this_row_offset] = (i-1)*N+j; // (j-1) + i * N; // (i-1)*N+j;
      values[this_row_offset] = -1;
      this_row_offset += 1;
    }
    if (j > 0) { // left neighbor
      col_indices[this_row_offset] = i*N+(j-1); // j + (i-1) * N;//i*N+(j-1);
      values[this_row_offset] = -1;
      this_row_offset += 1;
     }
    if (i < N-1) { // top node
      col_indices[this_row_offset] = (i+1)*N+j; // (j+1) + i * N; //i*N+(j+1);
      values[this_row_offset] = -1;
      this_row_offset += 1;
     }
    if (j < M-1) { // right node
      col_indices[this_row_offset] = i*N+(j+1); // j + (i+1) * N;// (i+1)*N+j;
      values[this_row_offset] = -1;
      this_row_offset += 1;
     }
  }
  // if(threadIdx.x == 0 && blockIdx.x == 0) values[0] = 100;
}
 
 
 
/** Implementation of the conjugate gradient algorithm.
 *
 *  The control flow is handled by the CPU.
 *  Only the individual operations (vector updates, dot products, sparse
 * matrix-vector product) are transferred to CUDA kernels.
 *
 *  The temporary arrays p, r, and Ap need to be allocated on the GPU for use
 * with CUDA. Modify as you see fit.
 */
void conjugate_gradient(int N, // number of unknows
                        int *csr_rowoffsets, int *csr_colindices,
                        double *csr_values, double *rhs, double *solution)
//, double *init_guess)   // feel free to add a nonzero initial guess as needed
{
  // initialize timer
  Timer timer;
 
  // clear solution vector (it may contain garbage values):
  std::fill(solution, solution + N, 0);
 
  // initialize work vectors:
  double alpha, beta;
  double *cuda_solution, *cuda_p, *cuda_r, *cuda_Ap, *cuda_scalar;
  hipMalloc(&cuda_p, sizeof(double) * N);
  hipMalloc(&cuda_r, sizeof(double) * N);
  hipMalloc(&cuda_Ap, sizeof(double) * N);
  hipMalloc(&cuda_solution, sizeof(double) * N);
  hipMalloc(&cuda_scalar, sizeof(double));
 
  hipMemcpy(cuda_p, rhs, sizeof(double) * N, hipMemcpyHostToDevice);
  hipMemcpy(cuda_r, rhs, sizeof(double) * N, hipMemcpyHostToDevice);
  hipMemcpy(cuda_solution, solution, sizeof(double) * N, hipMemcpyHostToDevice);
 
  const double zero = 0;
  double residual_norm_squared = 0;
  hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
  cuda_dot_product<<<512, 512>>>(N, cuda_r, cuda_r, cuda_scalar);
  hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);
 
  double initial_residual_squared = residual_norm_squared;
 
  int iters = 0;
  hipDeviceSynchronize();
  timer.reset();
  while (1) {
 
    // line 4: A*p:
    cuda_csr_matvec_product<<<512, 512>>>(N, csr_rowoffsets, csr_colindices, csr_values, cuda_p, cuda_Ap);
 
    // lines 5,6:
    hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
    cuda_dot_product<<<512, 512>>>(N, cuda_p, cuda_Ap, cuda_scalar);
    hipMemcpy(&alpha, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);
    alpha = residual_norm_squared / alpha;
 
    // line 7:
    cuda_vecadd<<<512, 512>>>(N, cuda_solution, cuda_p, alpha);
 
    // line 8:
    cuda_vecadd<<<512, 512>>>(N, cuda_r, cuda_Ap, -alpha);
 
    // line 9:
    beta = residual_norm_squared;
    hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
    cuda_dot_product<<<512, 512>>>(N, cuda_r, cuda_r, cuda_scalar);
    hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);
 
    // line 10:
    if (std::sqrt(residual_norm_squared / initial_residual_squared) < 1e-6) {
      break;
    }
 
    // line 11:
    beta = residual_norm_squared / beta;
 
    // line 12:
    cuda_vecadd2<<<512, 512>>>(N, cuda_p, cuda_r, beta);
 
    if (iters > 10000)
      break; // solver didn't converge
    ++iters;
  }
  hipMemcpy(solution, cuda_solution, sizeof(double) * N, hipMemcpyDeviceToHost);
 
  hipDeviceSynchronize();
  std::cout << "Time elapsed: " << timer.get() << " (" << timer.get() / iters << " per iteration)" << std::endl;
 
  if (iters > 10000)
    std::cout << "Conjugate Gradient did NOT converge within 10000 iterations"
              << std::endl;
  else
    std::cout << "Conjugate Gradient converged in " << iters << " iterations."
              << std::endl;
  
  std::ofstream out("heateq_data.csv");
  for (int i = 0; i < N; i++) {
      out << solution[i] <<',';
    }
  out.close();
 
 
  hipFree(cuda_p);
  hipFree(cuda_r);
  hipFree(cuda_Ap);
  hipFree(cuda_solution);
  hipFree(cuda_scalar);
}
 
/** Solve a system with `points_per_direction * points_per_direction` unknowns
 */
void solve_system(int points_per_direction) {
  Timer timer;
  int N = points_per_direction *
          points_per_direction; // number of unknows to solve for
  
  int my_N = points_per_direction;
  std::cout << "Solving Ax=b with " << N << " unknowns." << std::endl;
 
  //
  // Allocate CSR arrays.
  //
  // Note: Usually one does not know the number of nonzeros in the system matrix
  // a-priori.
  //       For this exercise, however, we know that there are at most 5 nonzeros
  //       per row in the system matrix, so we can allocate accordingly.
  //
  int *csr_rowoffsets = (int *)malloc(sizeof(double) * (N + 1));
  int *csr_colindices = (int *)malloc(sizeof(double) * 5 * N);
  double *csr_values = (double *)malloc(sizeof(double) * 5 * N);
 
  int *cuda_csr_rowoffsets, *cuda_csr_colindices;
  double *cuda_csr_values;
  //
  // fill CSR matrix with values
  //
  
  timer.reset();
  generate_fdm_laplace(points_per_direction, csr_rowoffsets, csr_colindices,
                       csr_values);
  printf("generate_fdm_laplace: %f\n", timer.get());
 
  
 
 
  // my generating fdm
  int *my_csr_rowoffsets = (int *)malloc(sizeof(double) * (N + 1));
  int *my_cuda_csr_rowoffsets; hipMalloc(&my_cuda_csr_rowoffsets, sizeof(double) * (N + 1));
  int *cuda_excl_scan_res; hipMalloc(&cuda_excl_scan_res, sizeof(double) * (N + 1));
  std::fill(my_csr_rowoffsets, my_csr_rowoffsets+N+1, 0);
  hipMemcpy(my_cuda_csr_rowoffsets, my_csr_rowoffsets, sizeof(double) * (N + 1), hipMemcpyHostToDevice);
  timer.reset();
  count_nnz<<<256,256>>>(my_cuda_csr_rowoffsets, my_N, my_N);
  printf("count_nnz: %f\n", timer.get());
 
 
  // hipMemcpy(my_csr_rowoffsets, my_cuda_csr_rowoffsets, sizeof(double) * N + 1, hipMemcpyDeviceToHost);
  //  for(int i = 0; i < N+1; i++) {
  //    printf("%i: %i\n", i, my_csr_rowoffsets[i]);
  //  }
  
  timer.get();
  exclusive_scan(my_cuda_csr_rowoffsets, cuda_excl_scan_res, N + 1);
  printf("excl scan: %f\n", timer.get());
  hipMemcpy(my_csr_rowoffsets, cuda_excl_scan_res, sizeof(double) * N + 1, hipMemcpyDeviceToHost);
 
 
  int *my_csr_col_indices = (int *)malloc(sizeof(double) * 5 * N);
  double *my_csr_values = (double *)malloc(sizeof(double) * 5 * N);
  // std::fill(my_csr_col_indices, my_csr_col_indices+5*N, 0);
  // std::fill(my_csr_values, my_csr_values+5*N, 0);
 
  int *my_cuda_csr_col_indices;
  double *my_cuda_csr_values;
  CUDA_ERRCHK(hipMalloc(&my_cuda_csr_col_indices, sizeof(double) * 5 * N));
  CUDA_ERRCHK(hipMalloc(&my_cuda_csr_values, sizeof(double) * 5 * N));
  // CUDA_ERRCHK(hipMemcpy(my_cuda_csr_col_indices, my_csr_col_indices, sizeof(double) * 5 * N, hipMemcpyHostToDevice));
  // CUDA_ERRCHK(hipMemcpy(my_cuda_csr_values, my_csr_values, sizeof(double) * 5 * N, hipMemcpyHostToDevice));
  // int *row_offsets, int N, int M, int *col_indices, double *values
  timer.reset();
  assembleA<<<256,256>>>(cuda_excl_scan_res, my_N, my_N, my_cuda_csr_col_indices, my_cuda_csr_values);
  printf("assemble: %f\n", timer.get());
  CUDA_ERRCHK(hipMemcpy(my_csr_col_indices, my_cuda_csr_col_indices, sizeof(double) * 5 * N, hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(my_csr_values, my_cuda_csr_values, sizeof(double) * 5 * N, hipMemcpyDeviceToHost));
  
  // for(int i = 0; i < 5 * N; i++) {
  //   printf("%i: %f\n", i, my_csr_values[i]);
  // }
 
 
 
 
  //
  // Allocate solution vector and right hand side:
  //
  double *solution = (double *)malloc(sizeof(double) * N);
  double *rhs = (double *)malloc(sizeof(double) * N);
  std::fill(rhs, rhs + N, 1);
 
  //
  // Allocate CUDA-arrays //
  //
  hipMalloc(&cuda_csr_rowoffsets, sizeof(double) * (N + 1));
  hipMalloc(&cuda_csr_colindices, sizeof(double) * 5 * N);
  hipMalloc(&cuda_csr_values, sizeof(double) * 5 * N);
  hipMemcpy(cuda_csr_rowoffsets, csr_rowoffsets, sizeof(double) * (N + 1), hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_colindices, csr_colindices, sizeof(double) * 5 * N,   hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_values,     csr_values,     sizeof(double) * 5 * N,   hipMemcpyHostToDevice);
  
  // for(int i = 0; i < 5 * N; i++) {
  //   printf("%i: %f, %f\n", i, my_csr_values[i], csr_values[i]);
  // }
 
  hipMemcpy(cuda_csr_rowoffsets, my_csr_rowoffsets, sizeof(double) * (N + 1), hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_colindices, my_csr_col_indices, sizeof(double) * 5 * N,   hipMemcpyHostToDevice);
  hipMemcpy(cuda_csr_values,     my_csr_values,     sizeof(double) * 5 * N,   hipMemcpyHostToDevice);
 
  //
  // Call Conjugate Gradient implementation with GPU arrays
  //
 
  timer.reset();
  conjugate_gradient(N, cuda_csr_rowoffsets, cuda_csr_colindices, cuda_csr_values, rhs, solution);
  printf("cg: %f\n", timer.get());
 
  //
  // Check for convergence:
  //
  double residual_norm = relative_residual(N, csr_rowoffsets, csr_colindices, csr_values, rhs, solution);
  std::cout << "Relative residual norm: " << residual_norm
            << " (should be smaller than 1e-6)" << std::endl;
 
  hipFree(cuda_csr_rowoffsets);
  hipFree(cuda_csr_colindices);
  hipFree(cuda_csr_values);
  free(solution);
  free(rhs);
  free(csr_rowoffsets);
  free(csr_colindices);
  free(csr_values);
}
 
int main() {
 
  solve_system(100); // solves a system with 100*100 unknowns
  CUDA_ERRCHK(hipDeviceReset());
  return EXIT_SUCCESS;
}