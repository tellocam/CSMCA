#include "hip/hip_runtime.h"
#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>


// result = (x, y)
__global__ void cuda_dot_product(int N, double *x, double *y, double *result)
{
  __shared__ double shared_mem[512];

  double dot = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    dot += x[i] * y[i];
  }

  shared_mem[threadIdx.x] = dot;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[threadIdx.x] += shared_mem[threadIdx.x + k];
    }
  }

  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}



/** Solve a system with `points_per_direction * points_per_direction` unknowns
 */
int main() {

  int N = 1000000;
  
  //
  // Allocate and initialize arrays on CPU
  //
  double *x = (double *)malloc(sizeof(double) * N);
  double *y = (double *)malloc(sizeof(double) * N);
  double alpha = 0;

  std::fill(x, x + N, 1);
  std::fill(y, y + N, 2);


  //
  // Allocate and initialize arrays on GPU
  //
  double *cuda_x;
  double *cuda_y;
  double *cuda_alpha;
  
  CUDA_ERRCHK(hipMalloc(&cuda_x, sizeof(double) * N));
  CUDA_ERRCHK(hipMalloc(&cuda_y, sizeof(double) * N));
  CUDA_ERRCHK(hipMalloc(&cuda_alpha, sizeof(double)));
  
  CUDA_ERRCHK(hipMemcpy(cuda_x, x, sizeof(double) * N, hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_y, y, sizeof(double) * N, hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice));
  
  cuda_dot_product<<<512, 512>>>(N, cuda_x, cuda_y, cuda_alpha);
  
  CUDA_ERRCHK(hipMemcpy(&alpha, cuda_alpha, sizeof(double), hipMemcpyDeviceToHost));

  std::cout << "Result of dot product: " << alpha << std::endl;

  //
  // Clean up
  //
  CUDA_ERRCHK(hipFree(cuda_x));
  CUDA_ERRCHK(hipFree(cuda_y));
  CUDA_ERRCHK(hipFree(cuda_alpha));
  free(x);
  free(y);

  return EXIT_SUCCESS;
}

