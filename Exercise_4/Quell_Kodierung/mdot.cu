#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <cmath>
#include <iostream>

int main(void)
{
    const size_t N = 100000;
    const size_t K = 16;

    //
    // Initialize CUBLAS:
    //
    std::cout << "Init CUBLAS..." << std::endl;
    hipblasHandle_t h;
    hipblasCreate(&h);


    //
    // allocate host memory:
    //
    std::cout << "Allocating host arrays..." << std::endl;
    double  *x = (double*)malloc(sizeof(double) * N);
    double **y = (double**)malloc(sizeof(double*) * K);
    for (size_t i=0; i<K; ++i) {
      y[i] = (double*)malloc(sizeof(double) * N);
    }
    double *results  = (double*)malloc(sizeof(double) * K);
    double *results2 = (double*)malloc(sizeof(double) * K);


    //
    // allocate device memory
    //
    std::cout << "Allocating CUDA arrays..." << std::endl;
    double *cuda_x; hipMalloc( (void **)(&cuda_x), sizeof(double)*N);
    double **cuda_y = (double**)malloc(sizeof(double*) * K);  // storing CUDA pointers on host!
    for (size_t i=0; i<K; ++i) {
      hipMalloc( (void **)(&cuda_y[i]), sizeof(double)*N);
    }

    //
    // fill host arrays with values
    //
    for (size_t j=0; j<N; ++j) {
      x[j] = 1 + j%K;
    }
    for (size_t i=0; i<K; ++i) {
      for (size_t j=0; j<N; ++j) {
        y[i][j] = 1 + rand() / (1.1 * RAND_MAX);
      }
    }

    //
    // Reference calculation on CPU:
    //
    for (size_t i=0; i<K; ++i) {
      results[i] = 0;
      results2[i] = 0;
      for (size_t j=0; j<N; ++j) {
        results[i] += x[j] * y[i][j];
      }
    }    
   
    //
    // Copy data to GPU
    //
    std::cout << "Copying data to GPU..." << std::endl;
    hipMemcpy(cuda_x, x, sizeof(double)*N, hipMemcpyHostToDevice);
    for (size_t i=0; i<K; ++i) {
      hipMemcpy(cuda_y[i], y[i], sizeof(double)*N, hipMemcpyHostToDevice);
    }


    //
    // Let CUBLAS do the work:
    //
    std::cout << "Running dot products with CUBLAS..." << std::endl;
    for (size_t i=0; i<K; ++i) {
      hipblasDdot(h, N, cuda_x, 1, cuda_y[i], 1, results2 + i);
    }

    //
    // Compare results
    //
    std::cout << "Copying results back to host..." << std::endl;
    for (size_t i=0; i<K; ++i) {
      std::cout << results[i] << " on CPU, " << results2[i] << " on GPU. Relative difference: " << fabs(results[i] - results2[i]) / results[i] << std::endl;
    }

    
    //
    // Clean up:
    //
    std::cout << "Cleaning up..." << std::endl;
    free(x);
    hipFree(cuda_x);

    for (size_t i=0; i<K; ++i) {
      free(y[i]);
      hipFree(cuda_y[i]);
    }
    free(y);
    free(cuda_y);

    free(results);
    free(results2);
 
    hipblasDestroy(h);
    return 0;
}
