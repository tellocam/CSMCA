#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"

const int threads_per_block = 256;
double dot_cpu(double *a, double *b, int N) {
   double product = 0;
   for (int i = 0; i < N; i++)
   product = product + a[i] * b[i];
   return product;
}
__global__ void dotVec_one(double *x, double *y, double *partial_z, int N) {
	__shared__ double temp_arr[threads_per_block];
	double thread_product = 0;
	unsigned int global_tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int local_tid  = threadIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;
	for (unsigned int i=global_tid; i<N; i+=total_threads) {
		thread_product += x[i] * y[i];
	}
	temp_arr[local_tid] = thread_product;
	for (unsigned int stride = blockDim.x/2; stride>0; stride/=2) {
		__syncthreads();
		if (threadIdx.x < stride) {
			temp_arr[threadIdx.x] += temp_arr[threadIdx.x + stride];
		}
	}
	if (threadIdx.x == 0) {
		partial_z[blockIdx.x] = temp_arr[0];
	}
}
__global__ void dotVec_two(double *partial_z) {
	for (int stride = blockDim.x/2; stride>0; stride/=2) {
		__syncthreads();
		if (threadIdx.x < stride)
			partial_z[threadIdx.x] += partial_z[threadIdx.x+stride];
	}
}
int main(void)
{
	// Task a //
	double *x, *y, *z;
	double *gpu_x, *gpu_y, *gpu_partial_z;
	Timer timer;
	int k = 0;
	int N_values_d[10] = { 100, 300, 1000, 3000, 10000, 30000, 100000, 300000, 1000000, 3000000 };
	printf("\nsize,time\n");
	while(k < 10) {
		int N = N_values_d[k];
		x = (double*)malloc(N*sizeof(double));
		y = (double*)malloc(N*sizeof(double));
		z = (double*)malloc(threads_per_block*sizeof(double));
		for (int i = 0; i < N; i++) {
			x[i] = 1.0;
			y[i] = 1.0;
		}
		hipMalloc(&gpu_x, N*sizeof(double)); 
		hipMalloc(&gpu_y, N*sizeof(double));
		hipMalloc(&gpu_partial_z, threads_per_block*sizeof(double));
		hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
		timer.reset();
		for (int n=0; n<5; n++) { 
			dotVec_one<<<256, threads_per_block>>>(gpu_x, gpu_y, gpu_partial_z, N);	
			hipDeviceSynchronize();
			dotVec_two<<<1, threads_per_block>>>(gpu_partial_z);
			hipMemcpy(z, gpu_partial_z, threads_per_block*sizeof(double), hipMemcpyDeviceToHost);
		}
		printf("%g,%g\n", z[0], 0.2*timer.get());
		hipFree(gpu_x);
		hipFree(gpu_y);
		hipFree(gpu_partial_z);
		free(x);
		free(y);
		free(z);
		k++;	
	}
	return EXIT_SUCCESS;
}
