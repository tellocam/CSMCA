#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1000000;

  float *x, *y, *d_x, *d_y;
  Timer timer;

  // Allocate host memory and initialize
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Allocate device memory and copy host data over
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // wait for previous operations to finish, then start timings
  hipDeviceSynchronize();
  timer.reset();

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  // wait for kernel to finish, then print elapsed time
  hipDeviceSynchronize();
  printf("Elapsed: %g\n", timer.get());

  // copy data back (implicit synchronization point)
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  // Numerical error check:
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  // tidy up host and device memory
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return EXIT_SUCCESS;
}

