#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"
#include <algorithm>
#include <vector>
// Kernel for Task 1b
__global__ void addVec_kth(double *x, double *y, double *z, int N, int k) {
	unsigned int total_threads = blockDim.x * gridDim.x;
	unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (k==0) {
		k = 1;
	}
	for (unsigned int i = global_tid; i<N-k; i += total_threads) {
		z[i+k] = x[i+k] + y[i+k];
	}
}

// findMedian function for any vector lenghts, source geeksforgeeks.com
double findMedian(std::vector<double> a,
                  int n)
{
    if (n % 2 == 0) {
        std::nth_element(a.begin(),
                    a.begin() + n / 2,
                    a.end());
        std::nth_element(a.begin(),
                    a.begin() + (n - 1) / 2,
                    a.end());
        return (double)(a[(n - 1) / 2]
                        + a[n / 2])
               / 2.0;
    }
    else {
        std::nth_element(a.begin(),
                    a.begin() + n / 2,
                    a.end());
        return (double)a[n / 2];
    }
}

int main(void)
{
	// Task 1b//
	double *x, *y, *z, *gpu_x, *gpu_y, *gpu_z;
	double eff_BW;
	Timer timer;
	int N = pow(10.0, 8.0);

	std::vector<int> k_values(64, 0);
	for(int i = 0; i<64; i++){
		k_values[i] = i;
	}
	std::vector<double> exec_timings = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	x = (double*)malloc(N*sizeof(double));
	y = (double*)malloc(N*sizeof(double));
	z = (double*)malloc(N*sizeof(double));

	for (int i = 0; i < N; i++) {
		x[i] = (double)(i);
		y[i] = (double)(N-i-1);
	}

	hipMalloc(&gpu_x, N*sizeof(double)); 
	hipMalloc(&gpu_y, N*sizeof(double));
	hipMalloc(&gpu_z, N*sizeof(double));
	hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < 64; i++) {
		for (int m = 0; m < 11; m++) {
			timer.reset();
			addVec_kth<<<256, 256>>>(gpu_x, gpu_y, gpu_z, N, k_values[i]);
			hipDeviceSynchronize();
			exec_timings[m] = timer.get();
		}
		eff_BW = 3 * floor((N - k_values[i])) * sizeof(double) * pow(10, -9) / findMedian(exec_timings, 10);
		printf("%d,%g\n", k_values[i], eff_BW);
	}

	hipFree(gpu_x);
	hipFree(gpu_y);
	hipFree(gpu_z);
	free(x);
	free(y);
	free(z);

	// // Task d //
	// int k = 0;
	// int N_values[10] = { 100, 300, 1000, 3000, 10000, 30000, 100000, 300000, 1000000, 3000000 };
	// printf("\nsize,time\n");
	// while(k < 10) {
	// 	float t_kernel=0;
	// 	int N = N_values[k];
	// 	x = (double*)malloc(N*sizeof(double));
	// 	y = (double*)malloc(N*sizeof(double));
	// 	z = (double*)malloc(N*sizeof(double));
	// 	for (int i = 0; i < N; i++) {
	// 		x[i] = (double)(i);
	// 		y[i] = (double)(N-i-1);
	// 	}
	// 	hipMalloc(&gpu_x, N*sizeof(double)); 
	// 	hipMalloc(&gpu_y, N*sizeof(double));
	// 	hipMalloc(&gpu_z, N*sizeof(double));
	// 	hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
	// 	hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
	// 	timer.reset();
	// 	for (int n=0; n<5; n++) {
	// 		addVec<<<256, 256>>>(gpu_x, gpu_y, gpu_z, N);
	// 		hipDeviceSynchronize();
	// 	}
	// 	t_kernel += timer.get();
	// 	printf("%d,%g\n", N, 0.2*t_kernel);
	// 	hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);
	// 	hipFree(gpu_x);
	// 	hipFree(gpu_y);
	// 	hipFree(gpu_z);
	// 	free(x);
	// 	free(y);
	// 	free(z);
	// 	k++;
	// }
	// // Task e //
	// N = 10000000;
	// k = 0;
	// int params[7] = { 16, 32, 64, 128, 256, 512, 1024};
	// printf("\nsqrt(threads),time\n");
	// while(k < 7) {
	// 	float t_kernel=0;
	// 	int param = params[k];
	// 	x = (double*)malloc(N*sizeof(double));
	// 	y = (double*)malloc(N*sizeof(double));
	// 	z = (double*)malloc(N*sizeof(double));
	// 	for (int i = 0; i < N; i++) {
	// 		x[i] = (double)(i);
	// 		y[i] = (double)(N-i-1);
	// 	}
	// 	hipMalloc(&gpu_x, N*sizeof(double)); 
	// 	hipMalloc(&gpu_y, N*sizeof(double));
	// 	hipMalloc(&gpu_z, N*sizeof(double));
	// 	hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
	// 	hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
	// 	timer.reset();
	// 	for (int n=0; n<5; n++) {
	// 		addVec<<<param, param>>>(gpu_x, gpu_y, gpu_z, N);
	// 		hipDeviceSynchronize();
	// 	}
	// 	t_kernel += timer.get();
	// 	printf("%d,%g\n", param, 0.2*t_kernel);
	// 	hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);
	// 	hipFree(gpu_x);
	// 	hipFree(gpu_y);
	// 	hipFree(gpu_z);
	// 	free(x);
	// 	free(y);
	// 	free(z);
	// 	k++;
	//}
	return EXIT_SUCCESS;
}

