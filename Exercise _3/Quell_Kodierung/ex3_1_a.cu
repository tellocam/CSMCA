#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"
__global__ void addVec_kth(double *x, double *y, double *z, int N) {
	unsigned int total_threads = blockDim.x * gridDim.x;
	unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int i = global_tid; i<N; i += total_threads) {
		z[i] = x[i] + y[i];
	}
}
int main(void)
{
	// Task a //
	double *x, *y, *z, *gpu_x, *gpu_y, *gpu_z;
	Timer timer;
	int N = pow(10.0, 8.0);
	x = (double*)malloc(N*sizeof(double));
	y = (double*)malloc(N*sizeof(double));
	z = (double*)malloc(N*sizeof(double));
	for (int i = 0; i < N; i++) {
		x[i] = (double)(i);
		y[i] = (double)(N-i-1);
	}
	hipMalloc(&gpu_x, N*sizeof(double)); 
	hipMalloc(&gpu_y, N*sizeof(double));
	hipMalloc(&gpu_z, N*sizeof(double));
	hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
	addVec_kth<<<256, 256>>>(gpu_x, gpu_y, gpu_z, N);
	hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(gpu_x);
	hipFree(gpu_y);
	hipFree(gpu_z);
	free(x);
	free(y);
	free(z);

	// Task d //
	int k = 0;
	int N_values[10] = { 100, 300, 1000, 3000, 10000, 30000, 100000, 300000, 1000000, 3000000 };
	printf("\nsize,time\n");
	while(k < 10) {
		float t_kernel=0;
		int N = N_values[k];
		x = (double*)malloc(N*sizeof(double));
		y = (double*)malloc(N*sizeof(double));
		z = (double*)malloc(N*sizeof(double));
		for (int i = 0; i < N; i++) {
			x[i] = (double)(i);
			y[i] = (double)(N-i-1);
		}
		hipMalloc(&gpu_x, N*sizeof(double)); 
		hipMalloc(&gpu_y, N*sizeof(double));
		hipMalloc(&gpu_z, N*sizeof(double));
		hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
		timer.reset();
		for (int n=0; n<5; n++) {
			addVec<<<256, 256>>>(gpu_x, gpu_y, gpu_z, N);
			hipDeviceSynchronize();
		}
		t_kernel += timer.get();
		printf("%d,%g\n", N, 0.2*t_kernel);
		hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);
		hipFree(gpu_x);
		hipFree(gpu_y);
		hipFree(gpu_z);
		free(x);
		free(y);
		free(z);
		k++;
	}
	// Task e //
	N = 10000000;
	k = 0;
	int params[7] = { 16, 32, 64, 128, 256, 512, 1024};
	printf("\nsqrt(threads),time\n");
	while(k < 7) {
		float t_kernel=0;
		int param = params[k];
		x = (double*)malloc(N*sizeof(double));
		y = (double*)malloc(N*sizeof(double));
		z = (double*)malloc(N*sizeof(double));
		for (int i = 0; i < N; i++) {
			x[i] = (double)(i);
			y[i] = (double)(N-i-1);
		}
		hipMalloc(&gpu_x, N*sizeof(double)); 
		hipMalloc(&gpu_y, N*sizeof(double));
		hipMalloc(&gpu_z, N*sizeof(double));
		hipMemcpy(gpu_x, x, N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(gpu_y, y, N*sizeof(double), hipMemcpyHostToDevice);
		timer.reset();
		for (int n=0; n<5; n++) {
			addVec<<<param, param>>>(gpu_x, gpu_y, gpu_z, N);
			hipDeviceSynchronize();
		}
		t_kernel += timer.get();
		printf("%d,%g\n", param, 0.2*t_kernel);
		hipMemcpy(z, gpu_z, N*sizeof(double), hipMemcpyDeviceToHost);
		hipFree(gpu_x);
		hipFree(gpu_y);
		hipFree(gpu_z);
		free(x);
		free(y);
		free(z);
		k++;
	}
	return EXIT_SUCCESS;
}

